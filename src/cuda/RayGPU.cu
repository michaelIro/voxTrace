#ifndef Ray_GPU_H
#define Ray_GPU_H

// X-Ray-Object for GPU
// Memory-Estimation: 1 instance of Ray approx. 133 byte => 10⁶ Rays approx. 133 MB 

#include <iostream>
#include <math.h> 
#include <hip/hip_runtime.h>




class RayGPU {
	// These Parameters are taken 1:1 from Shadow3
	float x0_, y0_, z0_;			// Start coordinates of X-Ray-photon
	float dirX_, dirY_, dirZ_;		// Coordinates of direction Vector of X-Ray-Photon
	float asX_, asY_, asZ_;			// sPolarization
	float apX_, apY_, apZ_;			// pPolarization
	bool flag_;						// Flag
	float k_;						// wave number
	int q_;							// ray index/number
	float opd_;						// optical path length 
	float fS_, fP_;					// Phases

	// Further Parameters needed for voxTrace
	int iaNum_;						// Number of interactions of this ray
	bool iaFlag_;					// Interaction Flag
	float prob_;					// Probability

	// int sum of sizes of all member variables
	size_t memory_size_= sizeof(float)* 17 + sizeof(int)*2 + sizeof(bool)*2 + sizeof(size_t);
		
  public:

	__host__ __device__ RayGPU() {};

  	__host__ __device__ RayGPU(float startX, float startY, float startZ, float dirX, float dirY, float dirZ, 
	  float asX, float asY, float asZ, bool flag, float k, int q, float opd, float fS, 
	  float fP, float apX, float apY, float apZ, float prob){
		x0_ = startX;
		y0_ = startY;
		z0_ = startZ;
	
		dirX_ = dirX; 	
		dirY_ = dirY;
		dirZ_ = dirZ;		

		asX_= asX;
		asY_= asY;
		asZ_= asZ;
		apX_= apX;
		apY_= apY;
		apZ_= apZ;

		flag_=flag;
		k_=k;
		q_=q;
		opd_=opd;
		fS_=fS;
		fP_=fP;
	
		iaNum_=0;
		iaFlag_=false;
		prob_= prob;
	};
	

	__host__ __device__ void setFlag(bool flag){flag_=flag;};
    __host__ __device__ void setStartCoordinates (float x, float y, float z) { x0_ = x; y0_ = y; z0_ = z;};
    __host__ __device__ void setEndCoordinates (float x, float y, float z) { dirX_ = x; dirY_ = y; dirZ_ = z;};
    __host__ __device__ void setSPol (float x, float y, float z) { asX_ = x; asY_ = y; asZ_ = z;};
    __host__ __device__ void setPPol (float x, float y, float z) { apX_ = x; apY_ = y; apZ_ = z;};
    __host__ __device__ void setEnergyKeV(float keV) {k_=keV*50677300.0;};
	__host__ __device__ void setIAFlag(bool iaFlag) {iaFlag_=iaFlag;};
	__host__ __device__ void setIANum(int iaNum) {iaNum_=iaNum;};

	__host__ __device__ void rotate(float phi, float theta){
			float diX = cos(theta)*cos(phi)*dirX_ - sin(phi)*dirY_ + sin(theta)*cos(phi)*dirZ_;
			float diY = cos(theta)*sin(phi)*dirX_ + cos(phi)*dirY_ + sin(theta)*sin(phi)*dirZ_;
			float diZ = -sin(theta)*dirX_+cos(theta)*dirZ_;
			dirX_=diX;
			dirY_=diY;
			dirZ_=diZ;
	}

	__host__ __device__ float getStartX() const {return x0_;};
	__host__ __device__ float getStartY() const {return y0_;};
	__host__ __device__ float getStartZ() const {return z0_;};
	__host__ __device__ float getDirX() const {return dirX_;};		
	__host__ __device__ float getDirY() const {return dirY_;};		
	__host__ __device__ float getDirZ() const {return dirZ_;};

	__host__ __device__ float getSPolX() const {return asX_;};
	__host__ __device__ float getSPolY() const {return asY_;};
	__host__ __device__ float getSPolZ() const {return asZ_;};
	__host__ __device__ float getPPolX() const {return apX_;};
	__host__ __device__ float getPPolY() const {return apY_;};
	__host__ __device__ float getPPolZ() const {return apZ_;};
	__host__ __device__ float getSPhase() const {return fS_;};
	__host__ __device__ float getPPhase() const {return fP_;};
		
	__host__ __device__ int getIndex() const {return q_;};
    __host__ __device__ bool getFlag() const {return flag_;};
    __host__ __device__ float getWaveNumber() const {return k_;};
    __host__ __device__ float getOpticalPath() const {return opd_;};
	__host__ __device__ float getEnergyEV() const {return (k_ / 50677300.0);};
    __host__ __device__ float getEnergyKeV() const {return (k_ / 50677300.0);};

	__host__ __device__ bool getIAFlag() const {return iaFlag_;};
	__host__ __device__ int getIANum() const {return iaNum_;};
	__host__ __device__ float getProb() const {return prob_;};

	__host__ void print() const { 
		std::cout << "Ray " << q_ << "\t Energy: \t" << getEnergyKeV() << " keV \t Memory-Size: " << memory_size_ << " Byte"<<std::endl;
		//std::cout << "Start: \t" << x0_ << "\t" << y0_ << "\t" << z0_ << std::endl;
		///std::cout << "Direction: \t" << dirX_ << "\t" << dirY_ << "\t" << dirZ_ << std::endl;
		//std::cout << "SPol: \t" << asX_ << "\t" << asY_ << "\t" << asZ_ << std::endl;
		//std::cout << "PPol: \t" << apX_ << "\t" << apY_ << "\t" << apZ_ << std::endl;
		//std::cout << "SPhase: \t" << fS_ << std::endl;
		//std::cout << "PPhase: \t" << fP_ << std::endl;
		//std::cout << "Flag: \t" << flag_ << std::endl;
		//std::cout << "WaveNumber: \t" << k_ << std::endl;
		//std::cout << "OpticalPath: \t" << opd_ << std::endl;
		//std::cout << "IAFlag: \t" << iaFlag_ << std::endl;
		//std::cout << "IANum: \t" << iaNum_ << std::endl;
		//std::cout << "Prob: \t" << prob_ << std::endl;
	};
};


#endif
#ifndef Ray_GPU_H
#define Ray_GPU_H

// X-Ray-Object for GPU
// Memory-Estimation: 1 instance of Ray approx. 133 byte => 10⁶ Rays approx. 133 MB 

#include <iostream>
#include <math.h> 
#include <hip/hip_runtime.h>




class RayGPU {
	// These Parameters are taken 1:1 from Shadow3
	float x0_, y0_, z0_;			// Start coordinates of X-Ray-photon
	float dirX_, dirY_, dirZ_;		// Coordinates of direction Vector of X-Ray-Photon
	float asX_, asY_, asZ_;			// sPolarization
	float apX_, apY_, apZ_;			// pPolarization
	bool flag_;						// Flag
	float k_;						// wave number
	int q_;							// ray index/number
	float opd_;						// optical path length 
	float fS_, fP_;					// Phases

	// Further Parameters needed for voxTrace
	int iaNum_;						// Number of interactions of this ray
	bool iaFlag_;					// Interaction Flag
	float prob_;					// Probability

	// int sum of sizes of all member variables
	size_t memory_size_= sizeof(float)* 17 + sizeof(int)*2 + sizeof(bool)*2 + sizeof(size_t);
		
  public:

	__host__ __device__ RayGPU() {};

  	__host__ __device__ RayGPU(float startX, float startY, float startZ, float dirX, float dirY, float dirZ, 
	  float asX, float asY, float asZ, bool flag, float k, int q, float opd, float fS, 
	  float fP, float apX, float apY, float apZ, float prob){
		x0_ = startX;
		y0_ = startY;
		z0_ = startZ;
	
		dirX_ = dirX; 	
		dirY_ = dirY;
		dirZ_ = dirZ;		

		asX_= asX;
		asY_= asY;
		asZ_= asZ;
		apX_= apX;
		apY_= apY;
		apZ_= apZ;

		flag_=flag;
		k_=k;
		q_=q;
		opd_=opd;
		fS_=fS;
		fP_=fP;
	
		iaNum_=0;
		iaFlag_=false;
		prob_= prob;
	};
	

	__host__ __device__ void setFlag(bool flag){flag_=flag;};
    __host__ __device__ void setStartCoordinates (float x, float y, float z) { x0_ = x; y0_ = y; z0_ = z;};
    __host__ __device__ void setEndCoordinates (float x, float y, float z) { dirX_ = x; dirY_ = y; dirZ_ = z;};
    __host__ __device__ void setSPol (float x, float y, float z) { asX_ = x; asY_ = y; asZ_ = z;};
    __host__ __device__ void setPPol (float x, float y, float z) { apX_ = x; apY_ = y; apZ_ = z;};
    __host__ __device__ void setEnergyKeV(float keV) {k_=keV*50677300.0;};
	__host__ __device__ void setIAFlag(bool iaFlag) {iaFlag_=iaFlag;};
	__host__ __device__ void setIANum(int iaNum) {iaNum_=iaNum;};

	__host__ __device__ void rotate(float phi, float theta){
			float diX = cos(theta)*cos(phi)*dirX_ - sin(phi)*dirY_ + sin(theta)*cos(phi)*dirZ_;
			float diY = cos(theta)*sin(phi)*dirX_ + cos(phi)*dirY_ + sin(theta)*sin(phi)*dirZ_;
			float diZ = -sin(theta)*dirX_+cos(theta)*dirZ_;
			dirX_=diX;
			dirY_=diY;
			dirZ_=diZ;
	}

	__host__ __device__ float getStartX() const {return x0_;};
	__host__ __device__ float getStartY() const {return y0_;};
	__host__ __device__ float getStartZ() const {return z0_;};
	__host__ __device__ float getDirX() const {return dirX_;};		
	__host__ __device__ float getDirY() const {return dirY_;};		
	__host__ __device__ float getDirZ() const {return dirZ_;};

	__host__ __device__ float getSPolX() const {return asX_;};
	__host__ __device__ float getSPolY() const {return asY_;};
	__host__ __device__ float getSPolZ() const {return asZ_;};
	__host__ __device__ float getPPolX() const {return apX_;};
	__host__ __device__ float getPPolY() const {return apY_;};
	__host__ __device__ float getPPolZ() const {return apZ_;};
	__host__ __device__ float getSPhase() const {return fS_;};
	__host__ __device__ float getPPhase() const {return fP_;};
		
	__host__ __device__ int getIndex() const {return q_;};
    __host__ __device__ bool getFlag() const {return flag_;};
    __host__ __device__ float getWaveNumber() const {return k_;};
    __host__ __device__ float getOpticalPath() const {return opd_;};
	__host__ __device__ float getEnergyEV() const {return (k_ / 50677300.0);};
    __host__ __device__ float getEnergyKeV() const {return (k_ / 50677300.0);};

	__host__ __device__ bool getIAFlag() const {return iaFlag_;};
	__host__ __device__ int getIANum() const {return iaNum_;};
	__host__ __device__ float getProb() const {return prob_;};

	__host__ __device__ void primaryTransform(float x0, float y0, float z0, float d, float alpha){

		float alpha_ = alpha / 180 * M_PI;

		float x0__ = x0 + getStartX();
		float y0__ = y0 - d * cos(alpha_) + cos(alpha_)*getStartY()-sin(alpha_)*getStartZ();
		float z0__ = z0 - d * sin(alpha_) + sin(alpha_)*getStartY()+cos(alpha_)*getStartZ();

		float xd__ = getDirX(); 
		float yd__ = cos(alpha_)*getDirY()-sin(alpha_)*getDirZ();
		float zd__ = sin(alpha_)*getDirY()+cos(alpha_)*getDirZ();

		setStartCoordinates(x0__,y0__,z0__);
		setEndCoordinates(xd__,yd__,zd__);
	}

	__host__ __device__ void secondaryTransform(float x0, float y0, float z0, float d, float beta){
		
		beta = beta / 180 * M_PI;

		if((getStartZ()>=0.0) && (getDirZ()<0.0)){

			float x0__ = getStartX() - x0;
			float y0__ = cos(beta)*(getStartY()-y0)-sin(beta)*(getStartZ()-z0);
			float z0__ = sin(beta)*(getStartY()-y0)+cos(beta)*(getStartZ()-z0);

			float xd__ = getDirX(); 
			float yd__ = cos(beta)*getDirY()-sin(beta)*getDirZ();
			float zd__ = sin(beta)*getDirY()+cos(beta)*getDirZ();


			float dfac_= (0.49-y0__) / yd__;
			float rin_= 0.1; //actually 0.095

			x0__= x0__ + dfac_ * xd__;
			y0__= 0.0;
			z0__= z0__ + dfac_ * zd__;

			//float r_spot_ = sqrt( (xd_*dfac_)*(xd_*dfac_) + (zd_*dfac_)*(zd_*dfac_));
			float r_spot_ = sqrt( (x0__*x0__) + (z0__*z0__) );

			if(r_spot_ < rin_){
				setStartCoordinates(x0__,y0__,z0__);
				setEndCoordinates(xd__,yd__,zd__);
				setIAFlag(true);
			}
			else{
				setIAFlag(false);
			}
		}
		else{
			setIAFlag(false);
		}

	}

	__host__ void print() const { 
		std::cout << "Ray " << q_ << "\t Energy: \t" << getEnergyKeV() << " keV \t Memory-Size: " << memory_size_ << " Byte"<<std::endl;
		//std::cout << "Start: \t" << x0_ << "\t" << y0_ << "\t" << z0_ << std::endl;
		///std::cout << "Direction: \t" << dirX_ << "\t" << dirY_ << "\t" << dirZ_ << std::endl;
		//std::cout << "SPol: \t" << asX_ << "\t" << asY_ << "\t" << asZ_ << std::endl;
		//std::cout << "PPol: \t" << apX_ << "\t" << apY_ << "\t" << apZ_ << std::endl;
		//std::cout << "SPhase: \t" << fS_ << std::endl;
		//std::cout << "PPhase: \t" << fP_ << std::endl;
		//std::cout << "Flag: \t" << flag_ << std::endl;
		//std::cout << "WaveNumber: \t" << k_ << std::endl;
		//std::cout << "OpticalPath: \t" << opd_ << std::endl;
		//std::cout << "IAFlag: \t" << iaFlag_ << std::endl;
		//std::cout << "IANum: \t" << iaNum_ << std::endl;
		//std::cout << "Prob: \t" << prob_ << std::endl;
	};
};


#endif
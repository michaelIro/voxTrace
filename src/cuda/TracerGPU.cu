#include "hip/hip_runtime.h"

//!  TracerGPU
#include "TracerGPU.cuh"


/*#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}*/

/** GPU kernel function to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) 
{
  hiprand_init(seed, 					      // the seed can be the same for each core, here we pass the time in from the CPU 
              blockIdx.x, 				  // the sequence number should be different for each core (unless you want all cores to get the same sequence of numbers for some reason - use thread id!
              0, 						        // the offset is how much extra we advance in the sequence for each call, can be 0 
              &states[blockIdx.x]);
}

/** GPU kernel function to create RayGPU objects from armadillo matrix 
__global__ void raygen(RayGPU* rays,  arma::Mat<double> beam_) 
{
      int i = blockIdx
		  rays[i] = RayGPU(beam_(i,0),beam_(i,1),beam_(i,2),beam_(i,3),beam_(i,4),beam_(i,5),beam_(i,6),beam_(i,7),beam_(i,8),beam_(i,9),
	  		beam_(i,10),beam_(i,11),beam_(i,12),beam_(i,13),beam_(i,14),beam_(i,15),beam_(i,16),beam_(i,17),beam_(i,18));
}*/ 

// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y)
{
  //for (int i = 0; i < n; i++)
    y[threadIdx.x] = x[threadIdx.x] + y[threadIdx.x];
}

// Kernel function to trace rays
__global__ void trace(RayGPU *rays, SampleGPU* sample, hiprandState_t *states)
{
	RayGPU*	currentRay = &rays[blockIdx.x];
  currentRay->primaryTransform(70.0, 70.0,0.0, 0.51, 45.0);
	VoxelGPU* currentVoxel = sample->findStartVoxel(currentRay);
	int nextVoxel = 13;	
	*currentRay = *TracerGPU::traceForward(currentRay, currentVoxel,&nextVoxel, sample,&states[blockIdx.x]);
  currentRay->secondaryTransform(70.0, 70.0,0.0, 0.49, 45.0);
	//printf("BlockIDX: %p ThreadIDX: %p\n",blockIdx.x,threadIdx.x);
	//printf("ThreadIDX %p\n",threadIdx.x);
}

// Kernel function to trace rays
__global__ void test(RayGPU *primary_rays,float *x, float *y, SampleGPU* sample, hiprandState_t *states)
{
    y[threadIdx.x] = sample->getOOBVoxel()->getMaterial()->CS_Tot(0.5);
    // generate a random number between 0 and 1 with cuRand
	//float r = hiprand_uniform (&states[threadIdx.x]);
    //y[threadIdx.x] = r;

	//traceForward(primary_rays[threadIdx.x], sample, states[threadIdx.x]);

	RayGPU*	currentRay = &primary_rays[threadIdx.x];
	VoxelGPU* currentVoxel = sample->findStartVoxel(currentRay);
	int nextVoxel = 13;	
	currentRay = TracerGPU::traceForward(currentRay, currentVoxel,&nextVoxel, sample,&states[threadIdx.x]);

	//RayGPU* aNewRay 
	//tracedRays[i++]=(*aNewRay);

    //printf("BlockIDX %f\n",r);
    //cuRandGenerateUniform(x, y, 1);
}

__device__  RayGPU* TracerGPU::traceForward(RayGPU* ray, VoxelGPU* currentVoxel, int* nextVoxel, SampleGPU *sample, hiprandState_t *localState){

	// Check if ray is already out of bounds of sample -> If so, no interaction is possible -> return ray.
	if(sample->isOOB(currentVoxel))
		return ray;
		
	float tIn;
	float rayEnergy = ray->getEnergyKeV();
	float muLin = currentVoxel->getMaterial()->CS_Tot_Lin(rayEnergy);
	float intersectionLength = currentVoxel->intersect(ray,nextVoxel,&tIn);
	float randomN = hiprand_uniform(localState);

	// Interaction in this Voxel?
	if(expf(-muLin*intersectionLength) < hiprand_uniform(localState)){

		// Selection of chemical Element to interact with	
		ChemElementGPU* interactingElement = currentVoxel->getMaterial()->getInteractingElement(rayEnergy,hiprand_uniform(localState));

		// Selection of interaction type
		int interactionType = interactingElement->getInteractionType(rayEnergy,hiprand_uniform(localState));

		if(interactionType == 0){ // Photo-Absorption
			
			// Selection of excited shell
			randomN = hiprand_uniform (localState);
			int myShell = interactingElement->getExcitedShell(rayEnergy,randomN);
			
			randomN = hiprand_uniform (localState);
			if(randomN > interactingElement->Fluor_Y(myShell)){ // Auger-Effect

				ray->setIANum(ray->getIANum()+1);
				ray->setIAFlag(true);
				ray->setFlag(false);
			}
			else{ // X-ray-Fluorescence

				randomN = hiprand_uniform (localState);
				int myLine = interactingElement->getTransition(myShell, randomN);

				randomN = hiprand_uniform (localState);
				float phi = 2*M_PI*randomN;

				randomN = hiprand_uniform (localState);
				float theta = acosf(2*randomN-1);

				randomN = hiprand_uniform (localState);
				float l = intersectionLength*randomN + tIn;

				float xNew = ray->getStartX()+ray->getDirX()*l;
				float yNew = ray->getStartY()+ray->getDirY()*l;
				float zNew = ray->getStartZ()+ray->getDirZ()*l;
				
				ray->rotate(phi,theta);
				ray->setStartCoordinates(xNew,yNew,zNew);
				ray->setEnergyKeV(interactingElement->Line_Energy(myLine));
				ray->setIANum(ray->getIANum()+1);
				ray->setIAFlag(true);
			}
		}
		else if(interactionType == 1){	// Rayleigh-Scattering TODO: Polarized-Unpolarized

			randomN = hiprand_uniform (localState);
			float phi = 2*M_PI*randomN;
				
			randomN = hiprand_uniform (localState);
			float theta = interactingElement->getThetaRayl(rayEnergy,randomN);	
	
			ray->rotate(phi,theta);

			randomN = hiprand_uniform (localState);
			float l = intersectionLength*randomN + tIn;

			float xNew = ray->getStartX()+ray->getDirX()*l;
			float yNew = ray->getStartY()+ray->getDirY()*l;
			float zNew = ray->getStartZ()+ray->getDirZ()*l;
			ray->setStartCoordinates(xNew,yNew,zNew);
			ray->setIANum(ray->getIANum()+1);
			ray->setIAFlag(true);
		}
		else if(interactionType == 2){ // Compton-Scattering

			randomN = hiprand_uniform (localState);
			float phi = 2*M_PI*randomN;

			randomN = hiprand_uniform (localState);
			float theta = interactingElement->getThetaCompt(rayEnergy,randomN);	

			ray->rotate(phi,theta);

			randomN = hiprand_uniform (localState);
			float l = intersectionLength*randomN + tIn;

			float xNew = ray->getStartX()+ray->getDirX()*l;
			float yNew = ray->getStartY()+ray->getDirY()*l;
			float zNew = ray->getStartZ()+ray->getDirZ()*l;
			ray->setStartCoordinates(xNew,yNew,zNew);
			ray->setIANum(ray->getIANum()+1);
			ray->setIAFlag(true);
		}
	}
	else{ // No interaction happening in this Voxel.
		currentVoxel =  currentVoxel->getNN(*nextVoxel);
		if(sample->isOOB(currentVoxel)) 
			ray->setFlag(false);
	}


	if((*ray).getFlag()){
		return traceForward(ray, currentVoxel,nextVoxel,sample,localState);
	}
	else {
		return ray;
	}
												
}

void TracerGPU::callTrace(){

  //int n_threads = 3;
  //int n_blocks = 3;

  int n_elements = 3;

  float x_=0.0, y_=0.0,z_=0.0;
  float xL_=150.0,yL_=150.0,zL_=5.0;
  float xLV_=15.0,yLV_=15.0,zLV_=0.5;
  int xN_ = (int)(xL_/xLV_)+1;
  int yN_ = (int)(yL_/yLV_)+1;
  int zN_ = (int)(zL_/zLV_)+1;
  //int voxN_= xN_*yN_*zN_;

  xLV_ = xL_/((float)(xN_-1));
  yLV_ = yL_/((float)(yN_-1));
  zLV_ = zL_/((float)(zN_-1));

  ChemElementGPU* elements;
  hipMallocManaged(&elements, sizeof(ChemElementGPU)*n_elements);

  float* weights;
  hipMallocManaged(&weights, sizeof(float)*n_elements*xN_*yN_*zN_);

  MaterialGPU* materials;
  hipMallocManaged(&materials, sizeof(MaterialGPU)*n_elements*xN_*yN_*zN_);
  
  VoxelGPU* voxels; 
  hipMallocManaged(&voxels, sizeof(VoxelGPU)*n_elements*xN_*yN_*zN_);

  VoxelGPU* oobVoxel; 
  hipMallocManaged(&oobVoxel, sizeof(VoxelGPU));

  SampleGPU* sample;
  hipMallocManaged(&sample, sizeof(SampleGPU));

  ChemElementGPU cu(29);
  ChemElementGPU sn(50);
  ChemElementGPU pb(82);

  elements[0] = cu;
  elements[1] = sn;
  elements[2] = pb;
  
  for(int i = 0; i < xN_; i++){
    for(int j = 0; j < yN_; j++){
      for(int k = 0; k < zN_; k++){
        weights[i*yN_*zN_*3+j*zN_*3+k*3+0] = 0.9;
        weights[i*yN_*zN_*3+j*zN_*3+k*3+1] = 0.05;
        weights[i*yN_*zN_*3+j*zN_*3+k*3+2] = 0.05;

        materials[i*yN_*zN_+j*zN_+k] = MaterialGPU(n_elements, elements, &weights[i*yN_*zN_*3+j*zN_*3+k*3+0]);
        voxels[i*yN_*zN_+j*zN_+k] = VoxelGPU(x_+i*xLV_, y_+j*yLV_, z_+k*zLV_, xLV_, yLV_, zLV_,&materials[i*yN_*zN_+j*zN_+k]);
      }
    }
  }

  *oobVoxel = VoxelGPU(-1.,-1.,-1.,-1.,-1.,-1.,&(materials[0]));
  *sample = SampleGPU(x_, y_,  z_, xL_,  yL_, zL_,  xLV_,  yLV_,  zLV_, xN_, yN_,  zN_, voxels, oobVoxel);

  //std::string path = "/tank/data/";
  std::string path = "/media/miro/Data/Documents/TU Wien/VSC-BEAM/";

  for (const auto & file : std::filesystem::directory_iterator(path)){
	  std::string pathname = file.path();
	  std::cout << pathname << std::endl;

	  arma::Mat<double> beam_;	// = new arma::Mat<double>();
    beam_.load(arma::hdf5_name(pathname, "my_data")); 
    std::cout << "beam_.n_rows: " << beam_.n_rows << std::endl;
    int N = beam_.n_rows;


    hiprandState_t* states;
    hipMallocManaged(&states, N*sizeof(hiprandState_t));

    init<<<N, 1>>>(time(0), states);

    RayGPU* rays;
    int success=0;

    hipMallocManaged(&rays, beam_.n_rows*sizeof(RayGPU));
    //hipMallocManaged(&s_rays, beam_.n_rows*sizeof(RayGPU));
	  clock_t begin = clock();

    for(int i = 0; i < beam_.n_rows; i++){
		  rays[i] = RayGPU(beam_(i,0),beam_(i,1),beam_(i,2),beam_(i,3),beam_(i,4),beam_(i,5),beam_(i,6),beam_(i,7),beam_(i,8),beam_(i,9),
	  		beam_(i,10),beam_(i,11),beam_(i,12),beam_(i,13),beam_(i,14),beam_(i,15),beam_(i,16),beam_(i,17),beam_(i,18));
    }

    clock_t middle = clock();


    //std::chrono::steady_clock::time_point t1_ = std::chrono::steady_clock::now();

    //std::cout << "READ FILE FOR: " << t1_-t0_ << std::endl;

    trace<<<N,1>>>(rays,sample, states);


    hipDeviceSynchronize();

    clock_t end = clock();
    double time_spent = (double)(end - middle) / CLOCKS_PER_SEC;
    double time_spent1 = (double)(middle - begin) / CLOCKS_PER_SEC;
    printf("Read: %f Trace: %f\n", time_spent, time_spent1);

	  for(int i = 0; i < beam_.n_rows; i++){
			//p_rays[i].secondaryTransform(70.0, 70.0,0.0, 0.49, 45.0);
      if(rays[i].getIAFlag())
        success++;
	  }
    std::cout << "sucess: " << success << std::endl;

	  hipFree(rays);
  }
 
  //free all memory cuda
  hipFree(elements);
  hipFree(materials);
  hipFree(voxels);
  hipFree(oobVoxel);
  hipFree(sample);
  hipFree(weights);
  
}

void TracerGPU::callAdd(){

  int N = 16;
  float *x, *y;

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    	x[i] = 1.0f;
    	y[i] = 2.0f;
  }

  std::cout<< "Before:"<<std::endl;
  for (int i = 0; i < N; i++) 
	  std::cout << "\t x[i]: " << x[i] << "\t y[i]:" << y[i] << std::endl;

  add<<<1,16>>>(16,x,y);

  hipDeviceSynchronize();

  std::cout<<std::endl<<std::endl<< "After:"<<std::endl;
  for (int i = 0; i < N; i++) 
	  std::cout << "\t x[i]: " << x[i] << "\t y[i]:" << y[i] << std::endl;

  //free all memory cuda
  hipFree(x);
  hipFree(y);
  
}

void TracerGPU::callTest(){
    //std::cout << "beam.ptr" << beam->memptr() << std::endl;

	//const arma::Mat<double> & beam_ = *beam;    
	arma::Mat<double> beam_;// = new arma::Mat<double>();
    
    beam_.load(arma::hdf5_name("/tank/data/PrimaryBeam-2-0.h5", "my_data")); 
    std::cout << "beam_.n_rows: " << beam_.n_rows << std::endl;
    //std::cout << "beam.ptr" << beam.memptr() << std::endl;


  RayGPU* primary_rays;
  //RayGPU* secondary_rays;

  hipMallocManaged(&primary_rays, beam_.n_rows*sizeof(RayGPU));
  //hipMallocManaged(&secondary_rays, beam_.n_rows*sizeof(RayGPU));

	std::cout<< "HIER" << beam_.n_rows<<std::endl;

		double  tetstl= beam_(0,0);
  for(int i = 0; i < beam_.n_rows; i++){
	primary_rays[i] = RayGPU(beam_(i,0),beam_(i,1),beam_(i,2),beam_(i,3),beam_(i,4),beam_(i,5),beam_(i,6),beam_(i,7),beam_(i,8),beam_(i,9),
	beam_(i,10),beam_(i,11),beam_(i,12),beam_(i,13),beam_(i,14),beam_(i,15),beam_(i,16),beam_(i,17),beam_(i,18));
  }

  for(int i = 0; i < 7; i++)
	primary_rays[i].print();


  // Test-Area
  int N = beam_.n_rows;
  float *x, *y;

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    	x[i] = 1.0f;
    	y[i] = 2.0f;
  }

  //std::cout<< "Before:"<<std::endl;
  //for (int i = 0; i < N; i++) 
//	  std::cout << "\t x[i]: " << x[i] << "\t y[i]:" << y[i] << std::endl;
  


  int n_elements = 3;
  //int n_threads = 3;

  float x_=0.0, y_=0.0,z_=0.0;
  float xL_=150.0,yL_=150.0,zL_=5.0;
  float xLV_=15.0,yLV_=15.0,zLV_=0.5;
  int xN_ = (int)(xL_/xLV_)+1;
  int yN_ = (int)(yL_/yLV_)+1;
  int zN_ = (int)(zL_/zLV_)+1;
  //int voxN_= xN_*yN_*zN_;

  xLV_ = xL_/((float)(xN_-1));
  yLV_ = yL_/((float)(yN_-1));
  zLV_ = zL_/((float)(zN_-1));

  ChemElementGPU* elements;
  hipMallocManaged(&elements, sizeof(ChemElementGPU)*n_elements);

  float* weights;
  hipMallocManaged(&weights, sizeof(float)*n_elements*xN_*yN_*zN_);

  MaterialGPU* materials;
  hipMallocManaged(&materials, sizeof(MaterialGPU)*n_elements*xN_*yN_*zN_);
  
  VoxelGPU* voxels; 
  hipMallocManaged(&voxels, sizeof(VoxelGPU)*n_elements*xN_*yN_*zN_);

  VoxelGPU* oobVoxel; 
  hipMallocManaged(&oobVoxel, sizeof(VoxelGPU));

  SampleGPU* sample;
  hipMallocManaged(&sample, sizeof(SampleGPU));

  hiprandState_t* states;
  hipMallocManaged(&states, N*sizeof(hiprandState_t));

  init<<<N, 1>>>(time(0), states);

  ChemElementGPU cu(29);
  ChemElementGPU sn(50);
  ChemElementGPU pb(82);

  elements[0] = cu;
  elements[1] = sn;
  elements[2] = pb;
  
  for(int i = 0; i < xN_; i++){
    for(int j = 0; j < yN_; j++){
      for(int k = 0; k < zN_; k++){
        weights[i*yN_*zN_*3+j*zN_*3+k*3+0] = i*yN_*zN_*3+j*zN_*3+k*3+0;
        weights[i*yN_*zN_*3+j*zN_*3+k*3+1] = i*yN_*zN_*3+j*zN_*3+k*3+1;
        weights[i*yN_*zN_*3+j*zN_*3+k*3+2] = i*yN_*zN_*3+j*zN_*3+k*3+2;

        materials[i*yN_*zN_+j*zN_+k] = MaterialGPU(n_elements, elements, &weights[i*yN_*zN_*3+j*zN_*3+k*3+0]);
        voxels[i*yN_*zN_+j*zN_+k] = VoxelGPU(x_+i*xLV_, y_+j*yLV_, z_+k*zLV_, xLV_, yLV_, zLV_,&materials[i*yN_*zN_+j*zN_+k]);
      }
    }
  }

  *oobVoxel = VoxelGPU(-1.,-1.,-1.,-1.,-1.,-1.,&(materials[0]));
  *sample = SampleGPU(x_, y_,  z_, xL_,  yL_, zL_,  xLV_,  yLV_,  zLV_, xN_, yN_,  zN_, voxels, oobVoxel);

  //test<<<1,16>>>(primary_rays,x,y,sample, states);
  trace<<<N,1>>>(primary_rays,sample, states);
  hipDeviceSynchronize();

  //std::cout<<std::endl<<std::endl<< "After:"<<std::endl;
  //for (int i = 0; i < N; i++) 
	//  std::cout << "\t x[i]: " << x[i] << "\t y[i]:" << y[i] << std::endl;

	for(int i = 0; i < beam_.n_rows; i++){
		if(primary_rays[i].getIANum() != 0)
			primary_rays[i].print();

	}


  //free all memory cuda
  hipFree(x);
  hipFree(y);
  hipFree(elements);
  hipFree(materials);
  hipFree(voxels);
  hipFree(oobVoxel);
  hipFree(sample);
  hipFree(weights);
  
}